#include "hip/hip_runtime.h"
#include "optixParams.h" // our launch params


extern "C" {
    __constant__ LaunchParams optixLaunchParams;
}

// ray types
enum { RAIDANCE = 0, SHADOW, RAY_TYPE_COUNT };

struct RadiancePRD {
    float3   emitted;
    float3   radiance;
    float3   attenuation;
    float3   origin;
    float3   direction;
    bool done;
    uint32_t seed;
    int32_t  countEmitted;
};

struct shadowPRD {
    float3 shadowAtt;
    uint32_t seed;
    int depth;
};




// -------------------------------------------------------

extern "C" __global__ void __closesthit__radiance() {

    const TriangleMeshSBTData& sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    RadiancePRD& prd = *(RadiancePRD*)getPRD<RadiancePRD>();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f - u - v) * sbtData.vertexD.normal[index.x]
        + u * sbtData.vertexD.normal[index.y]
        + v * sbtData.vertexD.normal[index.z];

    const float3 nn = normalize(make_float3(n));
    // intersection position
    const float3& rayDir = optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir;


    if (prd.countEmitted && length(sbtData.emission) != 0) {
        prd.emitted = sbtData.emission;
        return;
    }
    else
        prd.emitted = make_float3(0.0f);


    uint32_t seed = prd.seed;



    float3 diffuseColor;

    if (sbtData.hasTexture && sbtData.vertexD.texCoord0) {

        // compute pixel texture coordinate
        const float4 tc
            = (1.f - u - v) * sbtData.vertexD.texCoord0[index.x]
            + u * sbtData.vertexD.texCoord0[index.y]
            + v * sbtData.vertexD.texCoord0[index.z];
        // fetch texture value
        float4 fromTexture = tex2D<float4>(sbtData.texture, tc.x, tc.y);
        diffuseColor = make_float3(fromTexture);

    }
    else
        diffuseColor = sbtData.diffuse;




    const float r = rnd(seed);

    float continue_probability;
    if (optixLaunchParams.global->russian_roulette) {
        continue_probability = (diffuseColor.x + diffuseColor.y + diffuseColor.z) / 3;
    }
    else
    {
        continue_probability = 1.0f;
    }

    if (continue_probability <= r) {
        prd.done = true;
        return;
    }


    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    prd.seed = seed;


    // Assumir que a luz vem diretamente de cima
    float3 lightDir = make_float3(normalize(optixLaunchParams.global->lightDir));
    
        
    shadowPRD shadow_prd;
    shadow_prd.shadowAtt = make_float3(0);
    shadow_prd.depth = 0;
    {

        uint32_t u0, u1;
        packPointer(&shadow_prd, u0, u1);

        optixTrace(optixLaunchParams.traversable,
            pos,
            -lightDir,
            0.001f,         // tmin
            1e20f,          // tmax
            0.0f,                    // rayTime
            OptixVisibilityMask(1),
            OPTIX_RAY_FLAG_DISABLE_ANYHIT,
            SHADOW,      // SBT offset
            RAY_TYPE_COUNT,          // SBT stride
            SHADOW,      // missSBTIndex
            u0, u1);

    }

    {
        const float z1 = rnd(seed);
        const float z2 = rnd(seed);

        float3 w_in;
        cosine_sample_hemisphere(z1, z2, w_in);
        Onb onb(nn);
        onb.inverse_transform(w_in);
        prd.direction = w_in;
        prd.origin = pos;

        float spec_frac = pow(clamp(dot(normalize(optixGetWorldRayDirection()), normalize(reflect(-lightDir, nn))), 0.0, 1.0), sbtData.shininess);
        float3 brdf_spec = /*(sbtData.shininess + 2) / 2 **/ sbtData.specular * spec_frac;
        //Impedir que seja superior a 1 e verificar se est� na sombra
        brdf_spec *= shadow_prd.shadowAtt;
        float3 brdf = diffuseColor/**(1-spec_frac)*/ + brdf_spec;
        brdf = make_float3(min(1.0, brdf.x), min(1.0, brdf.y), min(1.0, brdf.z));
        prd.attenuation *= brdf;
        prd.countEmitted = false;
    }

    prd.radiance += shadow_prd.shadowAtt * optixLaunchParams.global->lightScale / continue_probability;
}


extern "C" __global__ void __anyhit__radiance() {

}


// miss sets the background color
extern "C" __global__ void __miss__radiance() {

    RadiancePRD& prd = *(RadiancePRD*)getPRD<RadiancePRD>();
    if (prd.countEmitted){
        prd.radiance = make_float3(0.529, 0.808, 0.922);
    }else{
        prd.radiance = make_float3(0);
    }
    prd.done = true;
}


// -----------------------------------------------
// Shadow rays

extern "C" __global__ void __closesthit__shadow() {

    shadowPRD& prd = *(shadowPRD*)getPRD<shadowPRD>();
    prd.shadowAtt = make_float3(0.0f);
}


// any hit for shadows
extern "C" __global__ void __anyhit__shadow() {

}


// miss for shadows
extern "C" __global__ void __miss__shadow() {

    shadowPRD& prd = *(shadowPRD*)getPRD<shadowPRD>();
    prd.shadowAtt = make_float3(1.0f);
}






// -----------------------------------------------
// Metal Phong rays

extern "C" __global__ void __closesthit__phong_metal() {


    const TriangleMeshSBTData& sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    RadiancePRD& prd = *(RadiancePRD*)getPRD<RadiancePRD>();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f - u - v) * sbtData.vertexD.normal[index.x]
        + u * sbtData.vertexD.normal[index.y]
        + v * sbtData.vertexD.normal[index.z];

    const float3 nn = normalize(make_float3(n));
    // intersection position
    const float3& rayDir = optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir;


    if (prd.countEmitted && length(sbtData.emission) != 0) {
        prd.emitted = sbtData.emission;
        return;
    }
    else
        prd.emitted = make_float3(0.0f);

    uint32_t seed = prd.seed;
    const float r = rnd(seed);
    if ((sbtData.specular.x + sbtData.specular.y + sbtData.specular.z) / 3 < r) {
        prd.done = true;
        return;
    }


    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    prd.seed = seed;


    const float glossiness = optixLaunchParams.global->glossiness;

    float3 reflectDir = reflect(optixGetWorldRayDirection(), nn);

    {
        const float z1 = rnd(seed);
        const float z2 = rnd(seed);

        float3 w_in;
        cosine_power_sample_hemisphere(z1, z2, w_in, glossiness);
        Onb onb(reflectDir);
        onb.inverse_transform(w_in);
        prd.direction = w_in;
        prd.origin = pos;


        prd.attenuation *= sbtData.specular;
        prd.countEmitted = false;
    }

}







// -----------------------------------------------
// Light material


extern "C" __global__ void __closesthit__light() {


    const TriangleMeshSBTData& sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    RadiancePRD& prd = *(RadiancePRD*)getPRD<RadiancePRD>();

    if (prd.countEmitted && length(sbtData.emission) != 0) {
        prd.emitted = sbtData.emission;
        return;
    }
    else
        prd.emitted = make_float3(0.0f);

    prd.countEmitted = false;
    prd.radiance = sbtData.diffuse * optixLaunchParams.global->lightScale;
}


extern "C" __global__ void __anyhit__light() {
}


extern "C" __global__ void __miss__light() {
    RadiancePRD& prd = *(RadiancePRD*)getPRD<RadiancePRD>();
    prd.radiance = make_float3(0.0f, 0.0f, 0.0f);
    prd.done = true;
}


extern "C" __global__ void __closesthit__light_shadow() {

    const TriangleMeshSBTData& sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f - u - v) * sbtData.vertexD.normal[index.x]
        + u * sbtData.vertexD.normal[index.y]
        + v * sbtData.vertexD.normal[index.z];

    float3 intersectionPoint = optixGetWorldRayOrigin() + optixGetRayTmax() * optixGetWorldRayDirection();
    float ndotl = max(0.0f, dot(normalize(make_float3(n)), -normalize(intersectionPoint - optixGetWorldRayOrigin())));
    shadowPRD& prd = *(shadowPRD*)getPRD<shadowPRD>();
    prd.shadowAtt = ndotl * sbtData.emission;
}



extern "C" __global__ void __anyhit__light_shadow() {
}


extern "C" __global__ void __miss__light_shadow() {
    shadowPRD& prd = *(shadowPRD*)getPRD<shadowPRD>();
    prd.shadowAtt = make_float3(0.0f);
}









// -----------------------------------------------
// Glass Phong rays


extern "C" __global__ void __closesthit__phong_glass() {


    const TriangleMeshSBTData& sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    RadiancePRD& prd = *(RadiancePRD*)getPRD<RadiancePRD>();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f - u - v) * sbtData.vertexD.normal[index.x]
        + u * sbtData.vertexD.normal[index.y]
        + v * sbtData.vertexD.normal[index.z];

    const float3 nn = normalize(make_float3(n));
    // intersection position
    const float3& rayDir = optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir;


    if (prd.countEmitted && length(sbtData.emission) != 0) {
        prd.emitted = sbtData.emission;
        return;
    }
    else
        prd.emitted = make_float3(0.0f);


    uint32_t seed = prd.seed;



    float3 diffuseColor;

    if (sbtData.hasTexture && sbtData.vertexD.texCoord0) {

        // compute pixel texture coordinate
        const float4 tc
            = (1.f - u - v) * sbtData.vertexD.texCoord0[index.x]
            + u * sbtData.vertexD.texCoord0[index.y]
            + v * sbtData.vertexD.texCoord0[index.z];
        // fetch texture value
        float4 fromTexture = tex2D<float4>(sbtData.texture, tc.x, tc.y);
        diffuseColor = make_float3(fromTexture);

    }
    else
        diffuseColor = sbtData.diffuse;


    const float r = rnd(seed);
    float continue_probability;
    if (optixLaunchParams.global->russian_roulette) {
        continue_probability = (diffuseColor.x + diffuseColor.y + diffuseColor.z) / 3;
    }
    else
    {
        continue_probability = 1.0f;
    }

    if (continue_probability < r) {
        prd.done = true;
        return;
    }


    {
        const float z1 = rnd(seed);
        prd.seed = seed;

        // new ray direction
        float3 rayDir = make_float3(0);

        const float3 normRayDir = optixGetWorldRayDirection();

        float indice_refracao_1;
        float indice_refracao_2;
        float3 r_normal;

        if (dot(normRayDir, nn) < 0) {
            indice_refracao_1 = 1;
            indice_refracao_2 = 1.5;
            r_normal = nn;
        }
        else {
            indice_refracao_1 = 1.5;
            indice_refracao_2 = 1;
            r_normal = -nn;
        }

        float costeta_i = dot(normalize(normRayDir), normalize(-r_normal));
        float costeta_t = sqrt(1 - (indice_refracao_1 / indice_refracao_2) * (indice_refracao_1 / indice_refracao_2) * (1 - (costeta_i) * (costeta_i)));

        float rs = (indice_refracao_2 * costeta_i - indice_refracao_1 * costeta_t) / (indice_refracao_2 * costeta_i + indice_refracao_1 * costeta_t) * (indice_refracao_2 * costeta_i - indice_refracao_1 * costeta_t) / (indice_refracao_2 * costeta_i + indice_refracao_1 * costeta_t);
        float rp = ((indice_refracao_2 * costeta_t - indice_refracao_1 * costeta_i) / (indice_refracao_2 * costeta_t + indice_refracao_1 * costeta_i)) * (indice_refracao_2 * costeta_t - indice_refracao_1 * costeta_i) / (indice_refracao_2 * costeta_t + indice_refracao_1 * costeta_i);
        float fr = (rs + rp) / 2.0;

        // Determinar se deve refratar
        if (z1 > fr) {
            rayDir = refract(normRayDir, r_normal, 1); //Vidro � apenas definido por um plano, por isso � apenas feita a transmissao raio sem desvios
        }
        // Caso a refra��o n�o seja poss�vel
        if (length(rayDir) == 0) {
            rayDir = reflect(normRayDir, r_normal);
        }

        prd.direction = rayDir;
        prd.origin = pos;

        prd.attenuation *= diffuseColor;
        //prd.countEmitted = false;
    }


    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    prd.seed = seed;



}



extern "C" __global__ void __anyhit__phong_glass() {

}


// miss sets the background color
extern "C" __global__ void __miss__phong_glass() {

    RadiancePRD& prd = *(RadiancePRD*)getPRD<RadiancePRD>();
    prd.radiance = make_float3(0.0f, 0.0f, 0.0f);
    prd.done = true;
}



// -----------------------------------------------
// Glass Shadow rays

extern "C" __global__ void __closesthit__shadow_glass() {

    shadowPRD& prd = *(shadowPRD*)getPRD<shadowPRD>();

    if (prd.depth > optixLaunchParams.global->shadowRays) {
        prd.shadowAtt = make_float3(0);
        return;
    }
    // ray payload
    shadowPRD afterPRD;
    afterPRD.shadowAtt = make_float3(1.0f);
    afterPRD.depth = prd.depth + 1;
    afterPRD.seed = prd.seed;
    uint32_t u0, u1;
    packPointer(&afterPRD, u0, u1);

    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * optixGetWorldRayDirection();


    const TriangleMeshSBTData& sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f - u - v) * sbtData.vertexD.normal[index.x]
        + u * sbtData.vertexD.normal[index.y]
        + v * sbtData.vertexD.normal[index.z];

    const float3 nn = normalize(make_float3(n));

    float indice_refracao_1;
    float indice_refracao_2;
    float3 r_normal;
    float3 normRayDir = optixGetWorldRayDirection();

    if (dot(normRayDir, nn) < 0) {
        indice_refracao_1 = 1;
        indice_refracao_2 = 1;
        r_normal = nn;

    }
    else {
        indice_refracao_1 = 1;
        indice_refracao_2 = 1;
        r_normal = -nn;
    }

    float3 rayDir = make_float3(0);

    uint32_t seed = prd.seed;

    const float z1 = rnd(seed);
    prd.seed = seed;

    float costeta_i = dot(normalize(normRayDir), normalize(-r_normal));
    float costeta_t = sqrt(1 - (indice_refracao_1 / indice_refracao_2) * (indice_refracao_1 / indice_refracao_2) * (1 - (costeta_i) * (costeta_i)));

    float rs = (indice_refracao_2 * costeta_i - indice_refracao_1 * costeta_t) / (indice_refracao_2 * costeta_i + indice_refracao_1 * costeta_t) * (indice_refracao_2 * costeta_i - indice_refracao_1 * costeta_t) / (indice_refracao_2 * costeta_i + indice_refracao_1 * costeta_t);
    float rp = ((indice_refracao_2 * costeta_t - indice_refracao_1 * costeta_i) / (indice_refracao_2 * costeta_t + indice_refracao_1 * costeta_i)) * (indice_refracao_2 * costeta_t - indice_refracao_1 * costeta_i) / (indice_refracao_2 * costeta_t + indice_refracao_1 * costeta_i);
    float fr = (rs + rp) / 2.0;

    // Determinar se deve refratar
    if (z1 > fr) {
        rayDir = refract(normRayDir, r_normal, indice_refracao_1 / indice_refracao_2);
    }
    // Caso a refra��o n�o seja poss�vel
    if (length(rayDir) == 0) {
        rayDir = reflect(normRayDir, r_normal);
    }

    // trace primary ray
    optixTrace(optixLaunchParams.traversable,
        pos,
        rayDir,
        0.001f,    // tmin
        1e20f,  // tmax
        0.0f,   // rayTime
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_DISABLE_ANYHIT, //OPTIX_RAY_FLAG_NONE,
        SHADOW,             // SBT offset
        RAY_TYPE_COUNT,     // SBT stride
        SHADOW,             // missSBTIndex 
        u0, u1);

    prd.shadowAtt = /*0.95f*/sbtData.diffuse * afterPRD.shadowAtt;
}


// any hit for shadows
extern "C" __global__ void __anyhit__shadow_glass() {

}


// miss for shadows
extern "C" __global__ void __miss__shadow_glass() {

    shadowPRD& prd = *(shadowPRD*)getPRD<shadowPRD>();
    prd.shadowAtt = make_float3(0.0f);
}










extern "C" __global__ void __closesthit__phong_alphaTrans()
{

    const TriangleMeshSBTData& sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    RadiancePRD& prd = *(RadiancePRD*)getPRD<RadiancePRD>();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f - u - v) * sbtData.vertexD.normal[index.x]
        + u * sbtData.vertexD.normal[index.y]
        + v * sbtData.vertexD.normal[index.z];

    const float3 nn = normalize(make_float3(n));
    // intersection position
    const float3& rayDir = optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir;


    if (prd.countEmitted && length(sbtData.emission) != 0) {
        prd.emitted = sbtData.emission;
        return;
    }
    else
        prd.emitted = make_float3(0.0f);


    uint32_t seed = prd.seed;



    float4 diffuseColor;

    if (sbtData.hasTexture && sbtData.vertexD.texCoord0) {

        // compute pixel texture coordinate
        const float4 tc
            = (1.f - u - v) * sbtData.vertexD.texCoord0[index.x]
            + u * sbtData.vertexD.texCoord0[index.y]
            + v * sbtData.vertexD.texCoord0[index.z];
        // fetch texture value
        float4 fromTexture = tex2D<float4>(sbtData.texture, tc.x, tc.y);
        diffuseColor = fromTexture;

    }
    else
        diffuseColor = make_float4(sbtData.diffuse, 1);




    const float r = rnd(seed);

    float continue_probability;
    if (optixLaunchParams.global->russian_roulette) {
        continue_probability = (diffuseColor.x + diffuseColor.y + diffuseColor.z) / 3;
    }
    else
    {
        continue_probability = 1.0f;
    }

    if (continue_probability <= r) {
        prd.done = true;
        return;
    }


    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    prd.seed = seed;


    // Assumir que a luz vem diretamente de cima
    float3 lightDir = make_float3(normalize(optixLaunchParams.global->lightDir));


    shadowPRD shadow_prd;
    shadow_prd.shadowAtt = make_float3(0);
    shadow_prd.depth = 0;
    if (diffuseColor.w > 0.5) {

        uint32_t u0, u1;
        packPointer(&shadow_prd, u0, u1);

        optixTrace(optixLaunchParams.traversable,
            pos,
            -lightDir,
            0.001f,         // tmin
            1e20f,          // tmax
            0.0f,                    // rayTime
            OptixVisibilityMask(1),
            OPTIX_RAY_FLAG_DISABLE_ANYHIT,
            SHADOW,      // SBT offset
            RAY_TYPE_COUNT,          // SBT stride
            SHADOW,      // missSBTIndex
            u0, u1);

        const float z1 = rnd(seed);
        const float z2 = rnd(seed);

        float3 w_in;
        cosine_sample_hemisphere(z1, z2, w_in);
        Onb onb(nn);
        onb.inverse_transform(w_in);
        prd.direction = w_in;
        prd.origin = pos;

        float spec_frac = pow(clamp(dot(normalize(optixGetWorldRayDirection()), normalize(reflect(-lightDir, nn))), 0.0, 1.0), sbtData.shininess);
        float3 brdf_spec = /*(sbtData.shininess + 2) / 2 **/ sbtData.specular * spec_frac;
        //Impedir que seja superior a 1 e verificar se est� na sombra
        brdf_spec *= shadow_prd.shadowAtt;
        float3 brdf = make_float3(diffuseColor) /** (1 - spec_frac)*/ + brdf_spec;
        brdf = make_float3(min(1.0, brdf.x), min(1.0, brdf.y), min(1.0, brdf.z));
        prd.attenuation *= brdf;
        prd.countEmitted = false;
        prd.radiance += shadow_prd.shadowAtt * optixLaunchParams.global->lightScale / continue_probability;
    }
    else {

        uint32_t u0, u1;
        packPointer(&prd, u0, u1);

        optixTrace(optixLaunchParams.traversable,
            pos,
            optixGetWorldRayDirection(),
            0.001f,    // tmin
            1e20f,  // tmax
            0.0f, OptixVisibilityMask(1),
            OPTIX_RAY_FLAG_NONE, RAIDANCE, RAY_TYPE_COUNT, RAIDANCE, u0, u1);
    }
}




extern "C" __global__ void __closesthit__shadow_alphaTrans() {

    const TriangleMeshSBTData& sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // intersection position
    const float3& rayDir = optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir;

    float4 diffuseColor;

    if (sbtData.hasTexture && sbtData.vertexD.texCoord0) {

        // compute pixel texture coordinate
        const float4 tc
            = (1.f - u - v) * sbtData.vertexD.texCoord0[index.x]
            + u * sbtData.vertexD.texCoord0[index.y]
            + v * sbtData.vertexD.texCoord0[index.z];
        // fetch texture value
        float4 fromTexture = tex2D<float4>(sbtData.texture, tc.x, tc.y);
        diffuseColor = fromTexture;

    }
    else
        diffuseColor = make_float4(sbtData.diffuse, 1);



    shadowPRD& prd = *(shadowPRD*)getPRD<shadowPRD>();

    if (diffuseColor.w > 0.5) {
        prd.shadowAtt = make_float3(0.0f);
    }
    else {

        uint32_t u0, u1;
        packPointer(&prd, u0, u1);

        optixTrace(optixLaunchParams.traversable,
            pos,
            rayDir,
            0.001f,         // tmin
            1e20f,          // tmax
            0.0f,                    // rayTime
            OptixVisibilityMask(1),
            OPTIX_RAY_FLAG_DISABLE_ANYHIT,
            SHADOW,      // SBT offset
            RAY_TYPE_COUNT,          // SBT stride
            SHADOW,      // missSBTIndex
            u0, u1);

    }

}






// -----------------------------------------------
// Primary Rays


extern "C" __global__ void __raygen__renderFrame() {

    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const auto& camera = optixLaunchParams.camera;

    const int& maxDepth = optixLaunchParams.frame.maxDepth;

    float squaredRaysPerPixel = float(optixLaunchParams.frame.raysPerPixel);
    float2 delta = make_float2(1.0f / squaredRaysPerPixel, 1.0f / squaredRaysPerPixel);

    float3 result = make_float3(0.0f);

    uint32_t seed = tea<4>(ix * optixGetLaunchDimensions().x + iy, optixLaunchParams.frame.frame);

    for (int i = 0; i < squaredRaysPerPixel; ++i) {
        for (int j = 0; j < squaredRaysPerPixel; ++j) {

            const float2 subpixel_jitter = make_float2(delta.x * (i + rnd(seed)), delta.y * (j + rnd(seed)));
            const float2 screen(make_float2(ix + subpixel_jitter.x, iy + subpixel_jitter.y)
                / make_float2(optixGetLaunchDimensions().x, optixGetLaunchDimensions().y) * 2.0 - 1.0);

            // note: nau already takes into account the field of view and ratio when computing 
            // camera horizontal and vertical
            float3 origin = camera.position;
            float3 rayDir = normalize(camera.direction
                + (screen.x) * camera.horizontal
                + (screen.y) * camera.vertical);

            RadiancePRD prd;
            prd.emitted = make_float3(0.f);
            prd.radiance = make_float3(0.f);
            prd.attenuation = make_float3(1.f);
            prd.countEmitted = true;
            prd.done = false;
            prd.seed = seed;

            uint32_t u0, u1;
            packPointer(&prd, u0, u1);

            for (int k = 0; k < maxDepth && !prd.done; ++k) {

                optixTrace(optixLaunchParams.traversable,
                    origin,
                    rayDir,
                    0.001f,    // tmin
                    1e20f,  // tmax
                    0.0f, OptixVisibilityMask(1),
                    OPTIX_RAY_FLAG_DISABLE_ANYHIT, RAIDANCE, RAY_TYPE_COUNT, RAIDANCE, u0, u1);

                result += prd.emitted;
                result += prd.radiance * prd.attenuation;

                origin = prd.origin;
                rayDir = prd.direction;

            }
        }
    }

    result = result / (squaredRaysPerPixel * squaredRaysPerPixel);
    float gamma = optixLaunchParams.global->gamma;
    // compute index
    const uint32_t fbIndex = ix + iy * optixGetLaunchDimensions().x;

    optixLaunchParams.global->accumBuffer[fbIndex] =
        (optixLaunchParams.global->accumBuffer[fbIndex] * optixLaunchParams.frame.subFrame +
            make_float4(result.x, result.y, result.z, 1)) / (optixLaunchParams.frame.subFrame + 1);


    float4 rgbaf = optixLaunchParams.global->accumBuffer[fbIndex];
    //convert float (0-1) to int (0-255)
    const int r = int(255.0f * min(1.0f, pow(rgbaf.x, 1 / gamma)));
    const int g = int(255.0f * min(1.0f, pow(rgbaf.y, 1 / gamma)));
    const int b = int(255.0f * min(1.0f, pow(rgbaf.z, 1 / gamma)));

    // convert to 32-bit rgba value 
    const uint32_t rgba = 0xff000000 | (r << 0) | (g << 8) | (b << 16);
    // write to output buffer
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
}



